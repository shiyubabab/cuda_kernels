#include <stdio.h>
#include <hip/hip_runtime.h>


typedef float data_t;
const int N = 32;

__global__ void sum(data_t *data){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int tx = threadIdx.x;

	printf("current thread global id is %d\n",idx);

	data[idx] += 1;
}

int main(void){
	int nbytes = sizeof(data_t) * N;
	data_t *d_mem,*h_mem;
	h_mem = (data_t *)malloc(nbytes);
	hipMalloc((void **)&d_mem,nbytes);

	for(int i = 0 ;i<N;i++) h_mem[i] = i;

	hipMemcpy(d_mem,h_mem,nbytes,hipMemcpyHostToDevice);

	sum<<<1,N>>>(d_mem);

	hipMemcpy(h_mem,d_mem,nbytes,hipMemcpyDeviceToHost);

	return 0;
}
