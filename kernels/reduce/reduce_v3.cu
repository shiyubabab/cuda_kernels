#include <stdio.h>
#include <hip/hip_runtime.h>


template<int blockSize>
__global__ void reduce_v3(const float *input,float *output){
	__shared__ float sdata[blockSize];
	int tid = threadIdx.x;
	int index = threadIdx.x + blockIdx.x * (blockSize*2);

	sdata[tid] = input[index]+input[index + blockSize];
	__syncthreads();

	for(unsigned int s = blockSize / 2;s>0;s>>=1){
		if(tid<s){
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if(tid == 0){
		output[blockIdx.x] = sdata[tid];
	}
}

int main(void){
	const int N = 25600000;
	float milliseconds = 0;

	const int blockSize = 256;
	int GridSize = (N + blockSize -1)/blockSize;
	float *h_mem,*d_mem;
	h_mem = (float *) malloc(sizeof(float)*N);
	hipMalloc((void **)&d_mem,sizeof(float)*N);

	float *h_ret,*d_ret;
	h_ret = (float *)malloc(sizeof(float)*GridSize);
	hipMalloc((void **)&d_ret,sizeof(float)*GridSize);

	for(int i = 0 ;i<N ;i++) h_mem[i] = 1.0f;

	hipMemcpy(d_mem,h_mem,sizeof(float)*N,hipMemcpyHostToDevice);

	dim3 Grid(GridSize);
	dim3 Block(blockSize/2);

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	reduce_v3<blockSize/2><<<Grid,Block>>>(d_mem,d_ret);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds,start,stop);

	hipMemcpy(h_ret,d_ret,GridSize*sizeof(float),hipMemcpyDeviceToHost);
	float res = 0;
	for(int i = 0; i <GridSize;i++){
		res += h_ret[i];
	}
	printf("The result is %f \n",res);
	printf("The reduce_v0 latency = %f ms \n",milliseconds);

	hipFree(d_mem);
	hipFree(d_ret);
	free(h_mem);
	free(h_ret);
	return 0;
}
