#include <stdio.h>
#include <hip/hip_runtime.h>


template<int blockSize>
__device__ void BlockSharedMemReduce(float *smem){
	if(blockSize>=1024){
		if(threadIdx.x<512){
			smem[threadIdx.x] += smem[threadIdx.x];
		}
		__syncthreads();
	}
	if(blockSize>=512){
		if(threadIdx.x<256){
			smem[threadIdx.x] += smem[threadIdx.x];
		}
		__syncthreads();
	}
	if(blockSize>=256){
		if(threadIdx.x<128){
			smem[threadIdx.x] += smem[threadIdx.x];
		}
		__syncthreads();
	}
	if(blockSize>=128){
		if(threadIdx.x<64){
			smem[threadIdx.x] += smem[threadIdx.x];
		}
		__syncthreads();
	}
	if(threadIdx.x < 32){
		volatile float *vshm = smem;
		float x = vshm[threadIdx.x];
		if(blockDim.x >= 64){
			x+=vshm[threadIdx.x+32];__syncwarp();
			vshm[threadIdx.x] = x; __syncwarp();
		}
		x+=vshm[threadIdx.x+16];__syncwarp();
		vshm[threadIdx.x] = x; __syncwarp();
		x+=vshm[threadIdx.x+8];__syncwarp();
		vshm[threadIdx.x] = x; __syncwarp();
		x+=vshm[threadIdx.x+4];__syncwarp();
		vshm[threadIdx.x] = x; __syncwarp();
		x+=vshm[threadIdx.x+2];__syncwarp();
		vshm[threadIdx.x] = x; __syncwarp();
		x+=vshm[threadIdx.x+1];__syncwarp();
		vshm[threadIdx.x] = x; __syncwarp();
	}
}

template<int blockSize>
__global__ void reduce_v5(const float *input,float *output){
	__shared__ float sdata[blockSize];
	int tid = threadIdx.x;
	int index = threadIdx.x + blockIdx.x * (blockSize*2);

	sdata[tid] = input[index]+input[index + blockSize];
	__syncthreads();

	BlockSharedMemReduce<blockSize>(sdata);

	if(tid == 0){
		output[blockIdx.x] = sdata[tid];
	}
}

int main(void){
	const int N = 25600000;
	float milliseconds = 0;

	const int blockSize = 256;
	int GridSize = (N + blockSize -1)/blockSize;
	float *h_mem,*d_mem;
	h_mem = (float *) malloc(sizeof(float)*N);
	hipMalloc((void **)&d_mem,sizeof(float)*N);

	float *h_ret,*d_ret;
	h_ret = (float *)malloc(sizeof(float)*GridSize);
	hipMalloc((void **)&d_ret,sizeof(float)*GridSize);

	for(int i = 0 ;i<N ;i++) h_mem[i] = 1.0f;

	hipMemcpy(d_mem,h_mem,sizeof(float)*N,hipMemcpyHostToDevice);

	dim3 Grid(GridSize);
	dim3 Block(blockSize/2);

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	reduce_v5<blockSize/2><<<Grid,Block>>>(d_mem,d_ret);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds,start,stop);

	hipMemcpy(h_ret,d_ret,GridSize*sizeof(float),hipMemcpyDeviceToHost);
	float res = 0;
	for(int i = 0; i <GridSize;i++){
		res += h_ret[i];
	}
	printf("The result is %f \n",res);
	printf("The reduce_v0 latency = %f ms \n",milliseconds);

	hipFree(d_mem);
	hipFree(d_ret);
	free(h_mem);
	free(h_ret);
	return 0;
}
