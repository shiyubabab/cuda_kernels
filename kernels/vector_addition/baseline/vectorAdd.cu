#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <iostream>
#include <vector>

__global__ void vectorAdd(const int *__restrict a,const int *__restrict b,
						  int *__restrict c,int N){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(tid < N) c[tid] = a[tid] + b[tid];
}

void verify_result(std::vector<int> &a,std::vector<int> &b,std::vector<int> &c){
	for(int i = 0;i<a.size();++i){
		assert(c[i] == a[i] + b[i]);
	}
}

int main(){
	const int N = 1<<16;
	const size_t bytes = sizeof(int) * N;

	std::vector<int> a;
	a.reserve(N);
	std::vector<int> b;
	b.reserve(N);
	std::vector<int> c;
	c.reserve(N);

	for(int i = 0;i<N;i++){
		a.push_back(rand()%100);
		b.push_back(rand()%100);
	}
	
	int *d_a,*d_b,*d_c;
	hipMalloc(&d_a,bytes);
	hipMalloc(&d_b,bytes);
	hipMalloc(&d_c,bytes);

	hipMemcpy(d_a,a.data(),bytes,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b.data(),bytes,hipMemcpyHostToDevice);

	int NUM_THREADS = 1 << 10; //Thread per CTA(1024)
	int NUM_BLOCKS = (N + NUM_THREAD -1) / NUM_THREADS;

	vectorAdd<<<NUM_BLOCKS,NUM_THREADS>>>(d_a,d_b,d_c,N);

	hipMemcpy(c.data(),d_c,bytes,hipMemcpyDeviceToHost);

	verify_result(a,b,c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	std::out << "SUCCESSFULLY " << std::endl;
	return 0;
}
