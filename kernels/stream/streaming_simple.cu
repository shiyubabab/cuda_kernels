

int main(){
	hipStream_t stream[3];
	float *A;
	float *d_A;
	for(int i = 0;i<3;i++) hipStreamCreate(&stream[i]);
	hipMalloc((void **)&d_A,30*sizeof(float));
	for(int i = 0;i<3;i++){
		hipMemcpyAsync(d_A + i*10*sizeof(float),A+i*10*sizeof(float),
						10*sizeof(float),hipMemcpyHostToDevice,stream[i]);
		float_add << <10,1,0,stream[i]> >>(d_A + i*10*sizeof(float));
		hipMemcpyAsync(d_A + i*10*sizeof(float),A+i*10*sizeof(float),
						10*sizeof(float),hipMemcpyDeviceToHost,stream[i]);
	}
	for(int i = 0;i<3;i++) hipStreamDestroy(stream[i]);
	hipHostFree(A);
	hipFree(A);
}
