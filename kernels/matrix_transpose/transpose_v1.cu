#include <stdio.h>
#include <hip/hip_runtime.h>


template<int blockSize>
__global__ void matrix_transpose_v0(const float *input,float *output,int X,int Y){
	const int ty = threadIdx.y,tx = threadIdx.x;

	int gtx = blockIdx.x * blockSize + tx;
	int gty = blockIdx.y * blockSize + ty;
	__shared__ float sdata[blockSize][blockSize];

	if(gty<Y && gtx<X){
		sdata[ty][tx] = input[gty * X + gtx];
	}
	__syncthreads();

	gtx = blockIdx.y * blockSize + tx;
	gty = blockIdx.x * blockSize + ty;
	if(gty<Y && gtx<X){
		output[gty * Y + gtx] = sdata[tx][ty];
	}
}


int main(void){
	float milliseconds = 0;
	const int Y = 2300;
	const int X = 1500;
	constexpr int N = X*Y;
	int matrix_size = N * sizeof(float);
	float *h_input,*d_input;
	float *h_output,*d_output;
	h_input = (float *)malloc(matrix_size);
	for(int i = 0;i<N;i++){
		h_input[i] = 2.0f;
	}
	hipMalloc((void **)&d_input,matrix_size);
	hipMemcpy(d_input,h_input,matrix_size,hipMemcpyHostToDevice);

	h_output = (float *)malloc(matrix_size);
	hipMalloc((void **)&d_output,matrix_size);
	
	const int blockSize = 16;
	int gridSize = (N+blockSize-1)/blockSize;
	dim3 block(blockSize,blockSize);
	dim3 grid(gridSize,gridSize);
	
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	matrix_transpose_v0<blockSize><<<grid,block>>>(d_input,d_output,X,Y);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds,start,stop);

	hipMemcpy(h_output,d_output,matrix_size,hipMemcpyDeviceToHost);
	printf("matrix transpose latency = %f ms \n",milliseconds);
	hipFree(d_input);
	hipFree(d_output);
	free(h_input);
	free(h_output);
	return 0;
}
