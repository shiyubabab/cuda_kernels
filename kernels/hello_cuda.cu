#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void hello_cude(){
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	printf("block id [%d], thread id [%d] and hello cuda!!!\n",blockIdx.x,idx);
}

int main(){
	hello_cuda<<<1,1>>>();
	hipDeviceSynchronize();
	return 0;
}
